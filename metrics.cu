#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <time.h>

#include "THC/THCTensor.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>

#include <stdexcept> 
#include <sstream>

//Code breaks with different values of this constant
#define BLOCK_DIM                     32
#define USE_BLOCK

#define cudaAssert(ans) { cudaAssert_((ans), __FILE__, __LINE__); }
inline void cudaAssert_(hipError_t code, const char *file, int line)
{

  if (code != hipSuccess) 
  {
    std::ostringstream out;
    out << "cuda error " << file << ":" << line << " " << hipGetErrorString(code);
    
    throw std::logic_error(out.str());
    
  }
}

/*
template<typename Op>
__global__ void cuDistanceSimple( float* x1, int size1, float* x2, int size2,  int dim,  float* distances, Op const &op)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    __shared__ int bx;
    __shared__ int by;
    
    bx = blockIdx.x * BLOCK_DIM;
    by = blockIdx.y * BLOCK_DIM;
       
    int inside1 = bx + tx < size1;
    int inside2 = by + ty < size2;
    
    float sum = 0.0;
    
      
    if (inside1 && inside2) {
      
      for (int k = 0; k < dim; ++k) {
        sum += op(x1[(bx + tx) * dim +  k], x2 [ (by + ty) * dim +  k]);
      }
    
      
    }

    if (inside1 && inside2) {
      distances[(bx + tx) * size2 + (by + ty)] = sum;
    }
}*/



template<typename Op>
__global__ void cuDistance( float* a, int size1, float* b, int size2,  int dim,  float* distances, Op const &op)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    __shared__ int bx, by;    
    
    bx = blockIdx.x * BLOCK_DIM;
    by = blockIdx.y * BLOCK_DIM;
    
    __shared__ float sa[BLOCK_DIM][BLOCK_DIM];
    __shared__ float sb[BLOCK_DIM][BLOCK_DIM];    
    
    int inside1 = (bx + tx < size1);   
    int inside2 = (by + ty < size2);   
    int inside3 = (by + tx < size2);
    
    int inside = inside1 && inside2;
    
    float sum = 0.0;
    
    for(int offk = 0; offk < dim; offk += BLOCK_DIM) {
         
      if (offk + ty < dim) {
         sa[tx][ty] = inside1 ? a[(bx + tx) * dim + ty + offk] : 0;
         sb[tx][ty] = inside3 ? b[(by + tx) * dim + ty + offk] : 0;
      } else {
        sa[tx][ty] = 0;
        sb[tx][ty] = 0;
      }
             
      __syncthreads();
      
      if(inside) {
        for (int k = 0; k < BLOCK_DIM; ++k) {
          sum += op(sa[tx][k], sb[ty][k]);
        }
      }
      
      __syncthreads();
    }

    if (inside) {
      distances[(bx + tx) * size2 + (by + ty)] = sum;
    }
}



struct LP {

  const float p;
  
  __host__ __device__ LP(const float &p) : p(p) { }
  
  __host__ __device__ float operator()(const float& x, const float& y) const {
    return pow(fabs(x - y), p);
  }
};


struct L1 {
   
  __host__ __device__ float operator()(const float& x, const float& y) const {
    return fabs(x - y);
  }
};


struct L2 {
   
  __host__ __device__ float operator()(const float& x, const float& y) const {
    float d = x - y;
    return d * d;
  }  
};



dim3 blocks(int ref_size, int query_size) {
  int bx = (ref_size  + BLOCK_DIM - 1)/BLOCK_DIM;
  int by = (query_size + BLOCK_DIM - 1)/BLOCK_DIM;

  return dim3(bx, by, 1);
}



template<typename Op>
void distance(float *ref, int ref_size,  float *query, int query_size, int dim, float *result, Op const &op) {
  dim3 grid = blocks(ref_size, query_size);
  dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);   
  
//   printf("dim = %d \n", dim);
  
  Op *deviceOp;
  hipMalloc((void**)&deviceOp, sizeof(Op));
  hipMemcpy((void*)deviceOp, (void*)&op, sizeof(Op), hipMemcpyHostToDevice);
  
  cuDistance<<<grid, threads>>>(ref, ref_size, query, query_size, dim, result, *deviceOp); 
  hipFree(deviceOp);
  
  cudaAssert(hipPeekAtLastError());  
}




void distanceL1(float *ref, int ref_size,  float *query, int query_size, int dim, float *result) {
  return distance(ref, ref_size, query, query_size, dim, result, L1());
}

void distanceL2(float *ref, int ref_size,  float *query, int query_size, int dim, float *result) {
  return distance(ref, ref_size, query, query_size, dim, result, L2());  
}


void distanceLP(float *ref, int ref_size,  float *query, int query_size, int dim,  float *result, float p) {
  return distance(ref, ref_size, query, query_size, dim, result, LP(p));
}


// int main(int argc, char **argv) {
//   
//   int dim = 250;
//   int n = 250;
//   
//   thrust::host_vector<float> x(n * dim);
//   for (int i = 0; i < n * dim; ++i) {
//     x[i] = i;
//   }
//   
//   thrust::device_vector<float> dx = x;
//   thrust::device_vector<float> result(n * n);
//   
//   float *xPtr = thrust::raw_pointer_cast(dx.data());
//   float *resultPtr = thrust::raw_pointer_cast(result.data());
//   
//   
//   thrust::device_vector<float> result1(n * n);
//   float *result1Ptr = thrust::raw_pointer_cast(result1.data());
// 
//   distanceL2(xPtr, n, xPtr, n, dim, resultPtr);
//   distanceSimpleL2(xPtr, n, xPtr, n, dim, result1Ptr);
//   
//   thrust::host_vector<float> r = result;
//   thrust::host_vector<float> r1 = result1;
//   
//   for (int i = 0; i < n * n; ++i) {
//     float err = fabs(r[i] - r1[i]);
//     if(err > 1) {
//       printf("fail %d: %f %f\n", i, r[i], r1[i]);  
//       break;
//     }
//   }
//   
// }

